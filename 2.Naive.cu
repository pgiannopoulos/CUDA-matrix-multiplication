#include "hip/hip_runtime.h"
﻿/*Απλός αλγόριθμος υπολογισμού γινομένου μητρώων, A^t * A = C, όπου Α (N*M), A^t (M*N) και C (M*M).*/
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

/*Μακρο-εντολή για έλεγχο λαθών*/
#define cudaCheckError() {                               \
  hipError_t e = hipGetLastError();                    \
  if (e != hipSuccess) {                                \
    printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
            hipGetErrorString(e));                      \
    exit(1);                                             \
  }                                                      \
}
/*Εκτύπωση των στοιχείων του μητρώου Α και των C_h, C που αποτελούν τα αποτελέσματα των γινομένων στην CPU και στην GPU αντίστοιχα.*/
void printMatrices(double *A_h, double *C_h, double *C, int M, int N)
{
	printf("Matrix A:\n");
	for (int i = 0; i < N*M; i++) {
		printf("%lf   ", A_h[i]);
		if ((i + 1) % M == 0)
			printf("\n");
	}
	printf("Multiplication Result on GPU:\n");
	for (int i = 0; i < M*M; i++) {
		printf("%lf   ", C_h[i]);
		if ((i + 1) % M == 0)
			printf("\n");
	}
	printf("Multiplication Result on CPU:\n");
	for (int i = 0; i < M*M; i++) {
		printf("%lf   ", C[i]);
		if ((i + 1) % M == 0)
			printf("\n");
	}
}
/*Υπολογισμός του ανάστροφου μητρώου Α.*/
double *transpose(double *A, int Α_rows, int Α_columns) {
	double *At_hranspose;
	int size = Α_rows * Α_columns * sizeof(double);
	At_hranspose = (double *)malloc(size);
	if ((At_hranspose == NULL)) {
		printf("Could not allocate memory.\n");
		exit(0);
	}
	int k = 0;
	for (int i = 0; i < Α_columns; i++)
		for (int j = 0; j < Α_rows; j++) {
			At_hranspose[k] = A[j*Α_columns + i];
			k++;
		}
	return At_hranspose;
}
/*Υπολογισμός του πολλαπλασιασμού 2 μητρώων στη CPU(host), A * B = C.*/
double *MatrixMulOnHost(double *A, double *B, int B_rows, int B_columns)
{
	double *C;
	int size = B_columns * B_columns * sizeof(double);
	C = (double *)malloc(size);
	if ((C == NULL)) {
		printf("Could not allocate memory.\n");
		exit(0);
	}
	for (int i = 0; i < B_columns; i++) {
		for (int j = 0; j < B_columns; j++) {
			double sum = 0;
			for (int k = 0; k < B_rows; k++)
			{
				double a = A[i*B_rows + k];
				double b = B[k*B_columns + j];
				sum += a*b;
			}
			C[i*B_columns + j] = sum;
		}
	}
	return C;
}

__global__ void naiveMul(double *A_d, double *C_d, int Acols, int Arows)
{
	//Υπολογισμός του δείκτη της γραμμής
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	//Υπολογισμός του δείκτη της στήλης
	int Col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((Row < Acols) && (Col < Acols)) {
		double Cvalue = 0.0;
		//Κάθε νήμα υπολογίζει ένα στοιχείο του κατά μπλοκ υπομητρώου
		for (int k = 0; k < Arows; k++)
			Cvalue += A_d[k * Acols + Row] * A_d[k * Acols + Col];

		C_d[Row * Acols + Col] = Cvalue;
	}
}

/*Υπολογισμός του γινομένου των μητρώων στη GPU και στη CPU για διασταύρωση των αποτελεσμάτων*/
int main()
{
	double *A_h, *C_h;	//Δείκτες για μνήμη στον host
	double *A_d, *C_d; 	//Δείκτες για μνήμη στο device
	double *At_h;	//Δείκτης για μνήμη στον host για τον ανάστροφο του Α
	double *C;		//Δείκτης για μνήμη στον host για τον υπολογισμό του γινομένου στη CPU
	int N, M;	//Διαστάσεις του μητρώου Α (Ν*Μ)
	int size_A, size_C;	//Μέγεθη των μητρώων στη μνήμη
	int i, j;	//Μετρητές

	//Χρήση του cudaEvent API ως μετρική απόδοσης
	hipEvent_t start, stop;
	hipEventCreate(&start);
	cudaCheckError();
	hipEventCreate(&stop);
	cudaCheckError();
	float milliseconds = 0;

	N = 1024;	//Ο αριθμός γραμμών του μητρώου Α 
	M = 1024;	//Ο αριθμός στηλών του μητρώου Α

	size_A = N * M * sizeof(double);
	size_C = M * M * sizeof(double);

	A_h = (double *)malloc(size_A);
	At_h = (double *)malloc(size_A);
	C_h = (double *)malloc(size_C);
	C = (double *)malloc(size_C);

	if ((A_h == NULL) || (At_h == NULL) || (C_h == NULL) || (C == NULL)) {
		printf("Could not allocate memory.\n");
		exit(0);
	}
	//Αρχικοποίηση του μητρώου Α με τυχαίες τιμές.
	for (i = 0; i < N; i++) {
		for (j = 0; j < M; j++) {
			A_h[i*M + j] = rand() / (double)RAND_MAX;
		}
	}

	// Δέσμευση μνήμης στο device για τα διανύσματα
	hipMalloc((void **)&A_d, size_A);
	cudaCheckError();
	//hipMalloc((void **)&B_d, size);
	hipMalloc((void **)&C_d, size_C);
	cudaCheckError();

	// Αντιγραφή A_h στο device
	hipMemcpy(A_d, A_h, size_A, hipMemcpyHostToDevice);
	cudaCheckError();

	//Κάθε block θα έχει διάσταση 32x32
	unsigned int BLOCK_SIZE_PER_DIM = 32;
	//Στρογγυλοποίηση προς τα πάνω για το πλήθος των block σε κάθε διάσταση
	unsigned int numBlocksX = (M - 1) / BLOCK_SIZE_PER_DIM + 1;
	unsigned int numBlocksY = (M - 1) / BLOCK_SIZE_PER_DIM + 1;
	//Ορισμός διαστάσεων πλέγματος
	dim3 dimGrid(numBlocksX, numBlocksY, 1);
	//Ορισμός διαστάσεων block
	dim3 dimBlock(BLOCK_SIZE_PER_DIM, BLOCK_SIZE_PER_DIM, 1);

	//Χρονομέτρηση της εκτέλεσης του πυρήνα
	hipEventRecord(start);
	cudaCheckError();
	naiveMul << < dimGrid, dimBlock >> > (A_d, C_d, M, N);
	cudaCheckError();
	hipEventRecord(stop);
	cudaCheckError();
	hipEventSynchronize(stop);
	cudaCheckError();
	hipEventElapsedTime(&milliseconds, start, stop);
	cudaCheckError();
	printf("Time Elapsed:%fms \n", milliseconds);

	//Αντιγραφή του αποτελέσματος C_d στον host
	hipMemcpy(C_h, C_d, size_C, hipMemcpyDeviceToHost);
	cudaCheckError();

	//Αποδέσμευση μνήμης στο device
	hipFree(A_d);
	cudaCheckError();
	hipFree(C_d);
	cudaCheckError();

	//Υπολογισμός του γινομένου στη CPU, Α_t * A_h = C.
	At_h = transpose(A_h, N, M);
	C = MatrixMulOnHost(At_h, A_h, N, M);

	//Εκτύπωση των μητρώων A_h, C_h και C.
	//printMatrices(A_h, C_h, C, M, N);

	//Αξιολόγηση της ορθότητας του πολλαπλασιασμού στη GPU.
	for (i = 0; i < M*M; i++) {
		if (C[i] - C_h[i] > 0.000001) {
			printf("The matrix multiplication on GPU was unsuccessful!\n");
			exit(0);
		}
	}
	printf("The matrix multiplication on GPU was successful!\n");
}